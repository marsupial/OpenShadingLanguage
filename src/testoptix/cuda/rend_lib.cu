#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optix_math.h>

#include "rend_lib.h"


rtDeclareVariable (uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable (uint2, launch_dim,   rtLaunchDim, );


// These functions are declared extern to prevent name mangling.
extern "C" {

    __device__
    void* closure_component_allot (void* pool, int id, size_t prim_size, const float3& w)
    { return 0; }


    __device__
    void* closure_mul_allot (void* pool, const float3& w, ClosureColor* c)
    { return 0; }


    __device__
    void* closure_mul_float_allot (void* pool, const float& w, ClosureColor* c)
    { return 0; }


    __device__
    void* closure_add_allot (void* pool, ClosureColor* a, ClosureColor* b)
    { return 0; }


    __device__
    void* osl_allocate_closure_component (void* sg_, int id, int size)
    { return 0; }


    __device__
    void* osl_allocate_weighted_closure_component (void* sg_, int id, int size, const float3* w)
    { return 0; }


    __device__
    void* osl_mul_closure_color (void* sg_, ClosureColor* a, float3* w)
    { return 0; }


    __device__
    void* osl_mul_closure_float (void* sg_, ClosureColor* a, float w)
    { return 0; }


    __device__
    void* osl_add_closure_closure (void* sg_, ClosureColor* a, ClosureColor* b)
    { return 0; }


    __device__
    int rend_get_userdata (char* name, void* data, int data_size)
    {
        return 0;
    }


    __device__
    int osl_bind_interpolated_param (void *sg_, const void *name, long long type,
                                     int userdata_has_derivs, void *userdata_data,
                                     int symbol_has_derivs, void *symbol_data,
                                     int symbol_data_size,
                                     char *userdata_initialized, int userdata_index)
    {
        int layer = 0;
        return rend_get_userdata ((char*)name, symbol_data, symbol_data_size);
    }


    __device__
    int osl_strlen_is (const char *str)
    {
        return DEVSTR(str).length();
    }


    __device__
    int osl_hash_is (const char *str)
    {
        return DEVSTR(str).hash();
    }


    __device__
    int osl_getchar_isi (const char *str, int index)
    {
        return (str && unsigned(index) < DEVSTR(str).length())
            ? str[index] : 0;
    }


    __device__
    void osl_printf (void* sg_, char* fmt_str, void* args)
    {
        printf (fmt_str, args);
    }

    __device__
    void* osl_get_texture_options (void *sg_)
    {
        return 0;
    }

    __device__
    int osl_texture (void *sg_, const char *name, void *handle,
             void *opt_, float s, float t,
             float dsdx, float dtdx, float dsdy, float dtdy,
             int chans, void *result, void *dresultdx, void *dresultdy,
             void *alpha, void *dalphadx, void *dalphady,
             void *ustring_errormessage)
    {
        if (!handle)
            return 0;
        int64_t texID = int64_t(handle);
        *((float3*)result) = make_float3(optix::rtTex2D<float4>(texID, s, t));
        return 1;
    }
}
