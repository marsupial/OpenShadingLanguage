#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optix_math.h>

#include "rend_lib.h"

#include <OSL/dual.h>
#include "../liboslexec/splineimpl.h"

rtBuffer<OSL_NAMESPACE::pvt::Spline::SplineBasis> gBasisSet;

rtDeclareVariable (uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable (uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable (char*, test_str_1, , );
rtDeclareVariable (char*, test_str_2, , );


// These functions are declared extern to prevent name mangling.
extern "C" {

    __device__
    void* closure_component_allot (void* pool, int id, size_t prim_size, const float3& w)
    {
        ((ClosureComponent*) pool)->id = id;
        ((ClosureComponent*) pool)->w  = w;

        size_t needed   = (sizeof(ClosureComponent) - sizeof(void*) + prim_size + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return (void*) &char_ptr[needed];
    }


    __device__
    void* closure_mul_allot (void* pool, const float3& w, ClosureColor* c)
    {
        ((ClosureMul*) pool)->id      = ClosureColor::MUL;
        ((ClosureMul*) pool)->weight  = w;
        ((ClosureMul*) pool)->closure = c;

        size_t needed   = (sizeof(ClosureMul) + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return &char_ptr[needed];
    }


    __device__
    void* closure_mul_float_allot (void* pool, const float& w, ClosureColor* c)
    {
        ((ClosureMul*) pool)->id       = ClosureColor::MUL;
        ((ClosureMul*) pool)->weight.x = w;
        ((ClosureMul*) pool)->weight.y = w;
        ((ClosureMul*) pool)->weight.z = w;
        ((ClosureMul*) pool)->closure  = c;

        size_t needed   = (sizeof(ClosureMul) + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return &char_ptr[needed];
    }


    __device__
    void* closure_add_allot (void* pool, ClosureColor* a, ClosureColor* b)
    {
        ((ClosureAdd*) pool)->id       = ClosureColor::ADD;
        ((ClosureAdd*) pool)->closureA = a;
        ((ClosureAdd*) pool)->closureB = b;

        size_t needed   = (sizeof(ClosureAdd) + 0x7) & ~0x7;
        char*  char_ptr = (char*) pool;

        return &char_ptr[needed];
    }


    __device__
    void* osl_allocate_closure_component (void* sg_, int id, int size)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        float3 w   = make_float3 (1.0f);
        void*  ret = sg_ptr->renderstate;

        size = max (4, size);

        sg_ptr->renderstate = closure_component_allot (sg_ptr->renderstate, id, size, w);

        return ret;
    }


    __device__
    void* osl_allocate_weighted_closure_component (void* sg_, int id, int size, const float3* w)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (w->x == 0.0f && w->y == 0.0f && w->z == 0.0f) {
            return NULL;
        }

        size = max (4, size);

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_component_allot (sg_ptr->renderstate, id, size, *w);

        return ret;
    }


    __device__
    void* osl_mul_closure_color (void* sg_, ClosureColor* a, float3* w)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (a == NULL) {
            return NULL;
        }

        if (w->x == 0.0f && w->y == 0.0f && w->z == 0.0f) {
            return NULL;
        }

        if (w->x == 1.0f && w->y == 1.0f && w->z == 1.0f) {
            return a;
        }

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_mul_allot (sg_ptr->renderstate, *w, a);

        return ret;
    }


    __device__
    void* osl_mul_closure_float (void* sg_, ClosureColor* a, float w)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (a == NULL || w == 0.0f) {
            return NULL;
        }

        if (w == 1.0f) {
            return a;
        }

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_mul_float_allot (sg_ptr->renderstate, w, a);

        return ret;
    }


    __device__
    void* osl_add_closure_closure (void* sg_, ClosureColor* a, ClosureColor* b)
    {
        ShaderGlobals* sg_ptr = (ShaderGlobals*) sg_;

        if (a == NULL) {
            return b;
        }

        if (b == NULL) {
            return a;
        }

        void* ret = sg_ptr->renderstate;
        sg_ptr->renderstate = closure_add_allot (sg_ptr->renderstate, a, b);

        return ret;
    }


    __device__
    int rend_get_userdata (char* name, void* data, int data_size,
                           long long type, int index)
    {
        // Perform a userdata lookup using the parameter name, type, and
        // userdata index. If there is a match, memcpy the value into data and
        // return 1.

        // TODO: This is temporary code for initial testing and demonstration.
        if (IS_STRING(type) && HDSTR(name) == HDSTR(test_str_1)) {
            memcpy (data, &test_str_2, 8);
            return 1;
        }

        return 0;
    }


    __device__
    int osl_bind_interpolated_param (void *sg_, const void *name, long long type,
                                     int userdata_has_derivs, void *userdata_data,
                                     int symbol_has_derivs, void *symbol_data,
                                     int symbol_data_size,
                                     char *userdata_initialized, int userdata_index)
    {
        int status = rend_get_userdata ((char*)name, userdata_data, symbol_data_size,
                                        type, userdata_index);
        return status;
    }


    __device__
    int osl_strlen_is (const char *str)
    {
        return HDSTR(str).length();
    }


    __device__
    int osl_hash_is (const char *str)
    {
        return HDSTR(str).hash();
    }


    __device__
    int osl_getchar_isi (const char *str, int index)
    {
        return (str && unsigned(index) < HDSTR(str).length())
            ? str[index] : 0;
    }


    __device__
    void osl_printf (void* sg_, char* fmt_str, void* args)
    {
        printf (fmt_str, args);
    }

    __device__
    void* osl_get_texture_options (void *sg_)
    {
        return 0;
    }

    __device__
    void osl_texture_set_interp_code(void *opt, int mode)
    {
        // ((TextureOpt *)opt)->interpmode = (TextureOpt::InterpMode)mode;
    }

    __device__
    void osl_texture_set_stwrap_code (void *opt, int mode)
    {
        //((TextureOpt *)opt)->swrap = (TextureOpt::Wrap)mode;
        //((TextureOpt *)opt)->twrap = (TextureOpt::Wrap)mode;
    }

    __device__
    int osl_texture (void *sg_, const char *name, void *handle,
             void *opt_, float s, float t,
             float dsdx, float dtdx, float dsdy, float dtdy,
             int chans, void *result, void *dresultdx, void *dresultdy,
             void *alpha, void *dalphadx, void *dalphady,
             void *ustring_errormessage)
    {
        if (!handle)
            return 0;
        int64_t texID = int64_t(handle);
        *((float3*)result) = make_float3(optix::rtTex2D<float4>(texID, s, t));
        return 1;
    }

    using namespace OSL_NAMESPACE::pvt;

    __device__
    void osl_spline_vfv(float3 *out, const char *spline_, float *x, 
                        float3 *knots, int knot_count, int knot_arraylen)
    {
        Spline::SplineInterp::create(HDSTR(spline_))
            .evaluate<float3, float, float3, float3, false>
                (*out, *x, knots, knot_count, knot_arraylen);

    }
/*
    __device__
    void osl_spline_fff(float *out, const char *spline_, float *x, 
                        float *knots, int knot_count, int knot_arraylen)
    {
        Spline::SplineInterp::create(HDSTR(spline_))
            .evaluate<float, float, float, float, false>
                (*out, *x, knots, knot_count, knot_arraylen);
    }

    __device__
    void osl_splineinverse_fff(float *out, const char *spline_, float *x, 
                               float *knots, int knot_count, int knot_arraylen)
    {
        // Version with no derivs
        Spline::SplineInterp::create(HDSTR(spline_))
            .inverse<float>
                (*out, *x, knots, knot_count, knot_arraylen);
    }

    __device__
    void osl_splineinverse_dffdf(float *out, const char *spline_, float *x, 
                                 float *knots, int knot_count, int knot_arraylen)
    {
        // Ignore knot derivs
        float outtmp = 0;
        osl_splineinverse_fff (&outtmp, spline_, x, knots, knot_count, knot_arraylen);
        *out = outtmp;
    }
*/
}
