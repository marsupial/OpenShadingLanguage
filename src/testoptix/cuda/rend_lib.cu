#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optix_math.h>

#include "rend_lib.h"
#include <OSL/dual.h>


rtDeclareVariable (uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable (uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable (char*, test_str_1, , );
rtDeclareVariable (char*, test_str_2, , );

__device__ constexpr int kNumSplineTypes = 6;
struct SplineBasis {
    int      basis_step;
    float    basis[4][4];
};
using CudaSplineBasis = SplineBasis[kNumSplineTypes];
struct SplineBasisArray {
    CudaSplineBasis splines;
};

//rtDeclareVariable(SplineBasisArray, gBasisSet, , );
rtBuffer<SplineBasis> gBasisSet;

namespace {

// ========================================================
//
// Interpolation bases for splines
//
// ========================================================

__device__ std::pair<const SplineBasis&, bool> getSplineBasis(const OSL::DeviceString& spline) {

/*
    if (spline == DeviceStrings::catmullrom)
        printf("CATMULL\n");
    if (spline == DeviceStrings::bezier)
        printf("BEZIER\n");
    if (spline == DeviceStrings::bspline)
        printf("BSPLINE\n");
    if (spline == DeviceStrings::hermite)
        printf("HERMITE\n");
    if (spline == DeviceStrings::constant)
        printf("CONSTANT\n");
    printf("LINEAR\n");
*/
    using namespace OSL_NAMESPACE;
    if (spline == DeviceStrings::catmullrom)
        return { gBasisSet[0], false };
    if (spline == DeviceStrings::bezier)
        return { gBasisSet[1], false };
    if (spline == DeviceStrings::bspline)
        return { gBasisSet[2], false };
    if (spline == DeviceStrings::hermite)
        return { gBasisSet[3], false };
    if (spline == DeviceStrings::constant)
        return { gBasisSet[5], true };

    return { gBasisSet[4], false };
}


// We need to know explicitly whether the knots have
// derivatives associated with them because of the way
// Dual2<T> forms of arrays are stored..  Arrays with 
// derivatives are stored:
//   T T T... T.dx T.dx T.dx... T.dy T.dy T.dy...
// This means, we need to explicitly construct the Dual2<T>
// form of the knots on the fly.
// if 'is_dual' == true, then OUTTYPE == Dual2<INTYPE>
// if 'is_dual' == false, then OUTTYPE == INTYPE

// This functor will extract a T or a Dual2<T> type from a VaryingRef array
template <class OUTTYPE, class INTYPE, bool is_dual>
struct extractValueFromArray
{
    __device__ OUTTYPE operator()(const INTYPE *value, int array_length, int idx);
};

template <class OUTTYPE, class INTYPE>
struct extractValueFromArray<OUTTYPE, INTYPE, true> 
{
    __device__ OUTTYPE operator()(const INTYPE *value, int array_length, int idx)
    {
        return OUTTYPE( value[idx + 0*array_length], 
                        value[idx + 1*array_length],
                        value[idx + 2*array_length] );
    }
};

template <class OUTTYPE, class INTYPE>
struct extractValueFromArray<OUTTYPE, INTYPE, false> 
{
    __device__ OUTTYPE operator()(const INTYPE *value, int array_length, int idx)
    {
        return OUTTYPE( value[idx] );
    }
};

template <class RTYPE, class XTYPE, class CTYPE, class KTYPE, bool knot_derivs >
__device__ void
spline_evaluate(const SplineBasis& spline, bool constant,
                RTYPE &result, XTYPE &xval, const KTYPE *knots,
                int knot_count, int knot_arraylen)
{
    using OIIO::clamp;
    XTYPE x = clamp(xval, XTYPE(0.0), XTYPE(1.0));
    int nsegs = ((knot_count - 4) / spline.basis_step) + 1;
    x = x*(float)nsegs;
    float seg_x = OSL::removeDerivatives(x);
    int segnum = (int)seg_x;
    if (segnum < 0)
        segnum = 0;
    if (segnum > (nsegs-1))
       segnum = nsegs-1;

    if (constant) {
        // Special case for "constant" basis
        RTYPE P = OSL::removeDerivatives (knots[segnum+1]);
        OSL::assignment (result, P);
        return;
    }

    // x is the position along segment 'segnum'
    x = x - float(segnum);
    int s = segnum * spline.basis_step;

    // create a functor so we can cleanly(!) extract
    // the knot elements
    extractValueFromArray<CTYPE, KTYPE, knot_derivs> myExtract;
    CTYPE P[4];
    for (int k = 0; k < 4; k++) {
        P[k] = myExtract(knots, knot_arraylen, s + k);
    }

    CTYPE tk[4];
    for (int k = 0; k < 4; k++) {
        tk[k] = spline.basis[k][0] * P[0] +
                spline.basis[k][1] * P[1] +
                spline.basis[k][2] * P[2] + 
                spline.basis[k][3] * P[3];
    }

    RTYPE tresult;
    // The following is what we want, but this gives me template errors
    // which I'm too lazy to decipher:
    //    tresult = ((tk[0]*x + tk[1])*x + tk[2])*x + tk[3];
    tresult = (tk[0]   * x + tk[1]);
    tresult = (tresult * x + tk[2]);
    tresult = (tresult * x + tk[3]);
    OSL::assignment(result, tresult);
}

}  // anonymous namespace

// These functions are declared extern to prevent name mangling.
extern "C" {

    __device__
    void* closure_component_allot (void* pool, int id, size_t prim_size, const float3& w)
    { return 0; }


    __device__
    void* closure_mul_allot (void* pool, const float3& w, ClosureColor* c)
    { return 0; }


    __device__
    void* closure_mul_float_allot (void* pool, const float& w, ClosureColor* c)
    { return 0; }


    __device__
    void* closure_add_allot (void* pool, ClosureColor* a, ClosureColor* b)
    { return 0; }


    __device__
    void* osl_allocate_closure_component (void* sg_, int id, int size)
    { return 0; }


    __device__
    void* osl_allocate_weighted_closure_component (void* sg_, int id, int size, const float3* w)
    { return 0; }


    __device__
    void* osl_mul_closure_color (void* sg_, ClosureColor* a, float3* w)
    { return 0; }


    __device__
    void* osl_mul_closure_float (void* sg_, ClosureColor* a, float w)
    { return 0; }


    __device__
    void* osl_add_closure_closure (void* sg_, ClosureColor* a, ClosureColor* b)
    { return 0; }


    __device__
    int rend_get_userdata (char* name, void* data, int data_size,
                           long long type, int index)
    {
        // Perform a userdata lookup using the parameter name, type, and
        // userdata index. If there is a match, memcpy the value into data and
        // return 1.

        // TODO: This is temporary code for initial testing and demonstration.
        if (IS_STRING(type) && HDSTR(name) == HDSTR(test_str_1)) {
            memcpy (data, &test_str_2, 8);
            return 1;
        }

        return 0;
    }


    __device__
    int osl_bind_interpolated_param (void *sg_, const void *name, long long type,
                                     int userdata_has_derivs, void *userdata_data,
                                     int symbol_has_derivs, void *symbol_data,
                                     int symbol_data_size,
                                     char *userdata_initialized, int userdata_index)
    {
        int status = rend_get_userdata ((char*)name, userdata_data, symbol_data_size,
                                        type, userdata_index);
        return status;
    }


    __device__
    int osl_strlen_is (const char *str)
    {
        return HDSTR(str).length();
    }


    __device__
    int osl_hash_is (const char *str)
    {
        return HDSTR(str).hash();
    }


    __device__
    int osl_getchar_isi (const char *str, int index)
    {
        return (str && unsigned(index) < HDSTR(str).length())
            ? str[index] : 0;
    }


    __device__
    void osl_printf (void* sg_, char* fmt_str, void* args)
    {
        printf (fmt_str, args);
    }

    __device__
    void* osl_get_texture_options (void *sg_)
    {
        return 0;
    }

    __device__
    void osl_texture_set_interp_code(void *opt, int mode)
    {
        // ((TextureOpt *)opt)->interpmode = (TextureOpt::InterpMode)mode;
    }

    __device__
    void osl_texture_set_stwrap_code (void *opt, int mode)
    {
        //((TextureOpt *)opt)->swrap = (TextureOpt::Wrap)mode;
        //((TextureOpt *)opt)->twrap = (TextureOpt::Wrap)mode;
    }

    __device__
    void osl_spline_vfv(float3 *out, const char *spline_, float *x, 
                        float3 *knots, int knot_count, int knot_arraylen)
    {
        const auto spline = getSplineBasis(HDSTR(spline_));
        spline_evaluate<float3, float, float3, float3, false>(spline.first, spline.second, *out,
                                                              *x, knots, knot_count, knot_arraylen);

    }

    __device__
    int osl_texture (void *sg_, const char *name, void *handle,
             void *opt_, float s, float t,
             float dsdx, float dtdx, float dsdy, float dtdy,
             int chans, void *result, void *dresultdx, void *dresultdy,
             void *alpha, void *dalphadx, void *dalphady,
             void *ustring_errormessage)
    {
        if (!handle)
            return 0;
        int64_t texID = int64_t(handle);
        *((float3*)result) = make_float3(optix::rtTex2D<float4>(texID, s, t));
        return 1;
    }
}
